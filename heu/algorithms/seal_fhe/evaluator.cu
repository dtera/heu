#include "hip/hip_runtime.h"
// Copyright 2024 Ant Group Co., Ltd.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "heu/algorithms/seal_fhe/evaluator.cuh"

#include <memory>
#include <vector>

#include "heu/spi/he/he_configs.h"
#include "heu/spi/utils/math_tool.h"

namespace heu::algos::seal_fhe {

Evaluator::Evaluator(const spi::Schema &schema, const SEALContext &context,
                     const seal_gpun::RelinKeys &relinKeys,
                     const seal_gpun::GaloisKeys &galoisKeys)
    : schema_(schema), relinKeys_(relinKeys), galoisKeys_(galoisKeys) {
  evaluator_ = std::make_shared<seal_gpun::Evaluator>(context);
}

Evaluator::Evaluator(const spi::Schema &schema,
                     const std::shared_ptr<seal_gpun::Evaluator> &evaluator,
                     const seal_gpun::RelinKeys &relinKeys,
                     const seal_gpun::GaloisKeys &galoisKeys)
    : schema_(schema),
      evaluator_(evaluator),
      relinKeys_(relinKeys),
      galoisKeys_(galoisKeys) {}

Plaintext Evaluator::Negate(const Plaintext &a) const { return Plaintext(); }

void Evaluator::NegateInplace(Plaintext *a) const {}

Ciphertext Evaluator::Negate(const Ciphertext &a) const {
  Ciphertext res;
  evaluator_->negate(a, res);
  return res;
}

void Evaluator::NegateInplace(Ciphertext *a) const {
  evaluator_->negateInplace(*a);
}

Plaintext Evaluator::Add(const Plaintext &a, const Plaintext &b) const {
  return Plaintext();
}

Ciphertext Evaluator::Add(const Ciphertext &a, const Plaintext &b) const {
  Ciphertext res;
  evaluator_->addPlain(a, b, res);
  return res;
}

Ciphertext Evaluator::Add(const Ciphertext &a, const Ciphertext &b) const {
  Ciphertext res;
  evaluator_->add(a, b, res);
  return res;
}

void Evaluator::AddInplace(Ciphertext *a, const Plaintext &b) const {
  evaluator_->addPlainInplace(*a, b);
}

void Evaluator::AddInplace(Ciphertext *a, const Ciphertext &b) const {
  evaluator_->addInplace(*a, b);
}

Plaintext Evaluator::Mul(const Plaintext &a, const Plaintext &b) const {
  return Plaintext();
}

Ciphertext Evaluator::Mul(const Ciphertext &a, const Plaintext &b) const {
  Ciphertext res;
  evaluator_->multiplyPlain(a, b, res);
  return res;
}

Ciphertext Evaluator::Mul(const Ciphertext &a, const Ciphertext &b) const {
  Ciphertext res;
  evaluator_->multiply(a, b, res);
  return res;
}

void Evaluator::MulInplace(Ciphertext *a, const Plaintext &b) const {
  evaluator_->multiplyPlainInplace(*a, b);
}

void Evaluator::MulInplace(Ciphertext *a, const Ciphertext &b) const {
  evaluator_->multiplyInplace(*a, b);
}

Plaintext Evaluator::Square(const Plaintext &a) const { return Mul(a, a); }

Ciphertext Evaluator::Square(const Ciphertext &a) const {
  Ciphertext res;
  evaluator_->square(a, res);
  return res;
}

void Evaluator::SquareInplace(Plaintext *a) const { *a = Square(*a); }

void Evaluator::SquareInplace(Ciphertext *a) const {
  evaluator_->squareInplace(*a);
}

template <typename T>
void Evaluator::DoPow(const T &a, int64_t exp, T *out) const {
  bool first = true;
  T s = a;
  while (exp != 0) {
    if (exp & 1) {
      if (first) {
        *out = s;
        first = false;
      } else {
        MulInplace(out, s);
      }
    }
    exp >>= 1;
    if (exp != 0) {
      MulInplace(&s, s);
    }
  }
}

Plaintext Evaluator::Pow(const Plaintext &a, int64_t exponent) const {
  Plaintext res;
  // DoPow(a, exponent, &res);
  return Plaintext();
}

Ciphertext Evaluator::Pow(const Ciphertext &a, int64_t exponent) const {
  Ciphertext res;
  DoPow(a, exponent, &res);
  return res;
}

void Evaluator::PowInplace(Plaintext *a, int64_t exponent) const {
  // DoPow(*a, exponent, a);
}

void Evaluator::PowInplace(Ciphertext *a, int64_t exponent) const {
  DoPow(*a, exponent, a);
}

void Evaluator::Randomize(Ciphertext *) const {
  // nothing to do
}

Ciphertext Evaluator::Relinearize(const Ciphertext &a) const {
  Ciphertext res;
  evaluator_->relinearize(a, relinKeys_, res);
  return res;
}

void Evaluator::RelinearizeInplace(Ciphertext *a) const {
  evaluator_->relinearizeInplace(*a, relinKeys_);
}

Ciphertext Evaluator::ModSwitch(const Ciphertext &a) const {
  return schema_ == spi::Schema::GPU_CKKS ? Rescale(a) : a;
}

void Evaluator::ModSwitchInplace(Ciphertext *a) const {
  if (schema_ == spi::Schema::GPU_CKKS) {
    RescaleInplace(a);
  }
}

Ciphertext Evaluator::Rescale(const Ciphertext &a) const {
  YACL_ENFORCE(schema_ == spi::Schema::GPU_CKKS,
               "Only gpu_ckks algo supports rescale");
  Ciphertext res;
  evaluator_->rescaleToNext(a, res);
  return res;
}

void Evaluator::RescaleInplace(Ciphertext *a) const {
  YACL_ENFORCE(schema_ == spi::Schema::GPU_CKKS,
               "Only gpu_ckks algo supports rescale");
  evaluator_->rescaleToNextInplace(*a);
}

Ciphertext Evaluator::SwapRows(const Ciphertext &a) const {
  YACL_ENFORCE(schema_ == spi::Schema::GPU_BFV,
               "Only bfv and bgv schema can swap rows");

  Ciphertext res;
  return res;
}

void Evaluator::SwapRowsInplace(Ciphertext *a) const {
  YACL_ENFORCE(schema_ == spi::Schema::GPU_BFV,
               "Only bfv and bgv schema can swap rows");
}

Ciphertext Evaluator::Conjugate(const Ciphertext &a) const {
  Ciphertext res;
  evaluator_->complexConjugate(a, galoisKeys_, res);
  return res;
}

void Evaluator::ConjugateInplace(Ciphertext *a) const {
  YACL_ENFORCE(schema_ == spi::Schema::GPU_CKKS,
               "Only ckks supports conjugate");
  evaluator_->complexConjugateInplace(*a, galoisKeys_);
}

// rotates the vector cyclically to the left (steps > 0) or to the right (steps
// < 0).
Ciphertext Evaluator::Rotate(const Ciphertext &a, int steps) const {
  Ciphertext res;
  evaluator_->rotateRows(a, steps, galoisKeys_, res);
  return res;
}

void Evaluator::RotateInplace(Ciphertext *a, int steps) const {
  evaluator_->rotateRowsInplace(*a, steps, galoisKeys_);
}

void Evaluator::BootstrapInplace(Ciphertext *) const {}

}  // namespace heu::algos::seal_fhe
