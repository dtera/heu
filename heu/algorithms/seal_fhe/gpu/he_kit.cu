#include "hip/hip_runtime.h"
// Copyright 2024 dterazhao Co., Ltd.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <variant>

#include "utils.h"

#include "heu/algorithms/seal_fhe/gpu/he_kit.cuh"

namespace heu::algos::seal_fhe::gpu {

static const std::string kLibName = "seal_fhe_gpu";  // do not change
static const std::map<int, std::vector<int>> poly_degree_bitsizes = {
    {1024, {20, 20}},
    {2048, {25, 25}},
    {4096, {60, 49}},
    {8192, {60, 49, 60}},
    {16384, {60, 40, 40, 40, 40, 60}},
    {32768, {60, 40, 40, 40, 40, 60}},
};  // do not change

std::shared_ptr<HeKit> HeKit::Create(
    const seal_gpun::EncryptionParameters &params) {
  return std::make_shared<HeKit>(params);
}

std::shared_ptr<HeKit> HeKit::Create(const seal_gpun::SchemeType &scheme,
                                     const std::size_t poly_modulus_degree,
                                     const std::vector<int> &bit_sizes,
                                     const double scale) {
  return std::make_shared<HeKit>(scheme, poly_modulus_degree, bit_sizes, scale);
}

std::shared_ptr<HeKit> HeKit::Create(const seal_gpun::SchemeType &scheme,
                                     const std::size_t poly_modulus_degree,
                                     const double scale) {
  return std::make_shared<HeKit>(scheme, poly_modulus_degree, scale);
}

HeKit::HeKit(const seal_gpun::EncryptionParameters &params) : scale_(1 << 6) {
  Init(params);
}

HeKit::HeKit(const seal_gpun::SchemeType &scheme,
             const std::size_t poly_modulus_degree,
             const std::vector<int> &bit_sizes, const double scale)
    : scale_(scale) {
  seal_gpun::KernelProvider::initialize();

  seal_gpu::EncryptionParameters params(scheme);
  params.setPolyModulusDegree(poly_modulus_degree);
  params.setCoeffModulus(
      seal_gpu::CoeffModulus::Create(poly_modulus_degree, bit_sizes));

  Init(params);
}

HeKit::HeKit(const seal_gpun::SchemeType &scheme,
             const std::size_t poly_modulus_degree, const double scale)
    : scale_(scale) {
  seal_gpun::KernelProvider::initialize();

  seal_gpu::EncryptionParameters params(scheme);
  params.setPolyModulusDegree(poly_modulus_degree);
  params.setCoeffModulus(seal_gpu::CoeffModulus::Create(
      poly_modulus_degree, poly_degree_bitsizes.at(poly_modulus_degree)));

  Init(params);
}

HeKit::~HeKit() {
  if (keygen_) delete keygen_;
  if (context_) delete context_;
  if (encryptor_) delete encryptor_;
  if (decryptor_) delete decryptor_;
  if (evaluator_) delete evaluator_;
  if (ckks_encoder_) delete ckks_encoder_;
  if (batch_encoder_) delete batch_encoder_;
}

std::string HeKit::GetLibraryName() const { return kLibName; }

void HeKit::Init(const seal_gpun::EncryptionParameters &params) {
  poly_modulus_degree_ = params.polyModulusDegree();
  slot_count_ = poly_modulus_degree_ / 2;
  context_ = new seal_gpun::SEALContext(params);
  keygen_ = new seal_gpun::KeyGenerator(*context_);
  pk_ = keygen_->createPublicKey();
  rlk_ = keygen_->createRelinKeys();
  glk_ = keygen_->createGaloisKeys();

  encryptor_ = new seal_gpun::Encryptor(*context_, pk_);
  decryptor_ = new seal_gpun::Decryptor(*context_, keygen_->secretKey());
  evaluator_ = new seal_gpun::Evaluator(*context_);
  if (params.scheme() == seal_gpun::SchemeType::ckks) {
    ckks_encoder_ = new seal_gpun::CKKSEncoder(*context_);
    // encoder_ = ckks_encoder_;
  } else {
    batch_encoder_ = new seal_gpun::BatchEncoder(*context_);
    // encoder_ = batch_encoder_;
  }
}

//==========================fhe_gpu operation bigin==========================
void HeKit::Encode(const int64_t m, seal_gpun::Plaintext &out) {
  ckks_encoder_->encode(m, out);
}

void HeKit::Encode(const double m, seal_gpun::Plaintext &out) {
  ckks_encoder_->encode(m, scale_, out);
}

void HeKit::Encode(const std::vector<double> &ms, seal_gpun::Plaintext &out) {
  ckks_encoder_->encodePolynomial(ms, scale_, out);
}

void HeKit::Encode(const std::vector<std::complex<double>> &ms,
                   seal_gpun::Plaintext &out) {
  ckks_encoder_->encode(ms, scale_, out);
}

template <typename T,
          typename std::enable_if_t<std::is_arithmetic_v<T>, int64_t>>
void HeKit::Encode(const std::vector<T> &ms,
                   std::vector<seal_gpun::Plaintext> &out, bool async,
                   int32_t n_threads) {
  if (async) {
    ParallelFor(ms.size(), n_threads, [&](int i) { Encode(ms[i], out[i]); });
  } else {
    for (int i = 0; i < ms.size(); i++) {
      Encode(ms[i], out[i]);
    }
  }
}

double HeKit::Decode(seal_gpun::Plaintext &pt) {
  std::vector<double> out(1);
  ckks_encoder_->decodePolynomial(pt, out);
  return out[0];
}

void HeKit::Decode(const seal_gpun::Plaintext &pt, std::vector<double> &out) {
  ckks_encoder_->decodePolynomial(pt, out);
}

void HeKit::Decode(const seal_gpun::Plaintext &pt,
                   std::vector<std::complex<double>> &out) {
  ckks_encoder_->decode(pt, out);
}

void HeKit::Encrypt(const seal_gpun::Plaintext &pt,
                    seal_gpun::Ciphertext &out) {
  encryptor_->encrypt(pt, out);
}

template <typename T,
          typename std::enable_if_t<std::is_arithmetic_v<T>, int64_t>>
void HeKit::Encrypt(const T m, seal_gpun::Ciphertext &out) {
  seal_gpun::Plaintext pt;
  Encode(m, pt);
  encryptor_->encrypt(pt, out);
}

template <typename T>
void HeKit::Encrypt(const std::vector<T> &pts,
                    std::vector<seal_gpun::Ciphertext> &out, bool async,
                    int32_t n_threads) {
  if (async) {
    ParallelFor(pts.size(), n_threads, [&](int i) { Encrypt(pts[i], out[i]); });
  } else {
    for (int i = 0; i < pts.size(); i++) {
      Encrypt(pts[i], out[i]);
    }
  }
}

void HeKit::Decrypt(const seal_gpun::Ciphertext &ct,
                    seal_gpun::Plaintext &out) {
  decryptor_->decrypt(ct, out);
}

void HeKit::Decrypt(const std::vector<seal_gpun::Ciphertext> &cts,
                    std::vector<seal_gpun::Plaintext> &out, bool async,
                    int32_t n_threads) {
  if (async) {
    ParallelFor(cts.size(), n_threads, [&](int i) { Decrypt(cts[i], out[i]); });
  } else {
    for (int i = 0; i < cts.size(); i++) {
      Decrypt(cts[i], out[i]);
    }
  }
}

void HeKit::Eval(
    const std::vector<seal_gpun::Ciphertext> &cts1,
    const std::vector<seal_gpun::Ciphertext> &cts2,
    std::vector<seal_gpun::Ciphertext> &out,
    std::function<void(const seal_gpun::Ciphertext &,
                       const seal_gpun::Ciphertext &, seal_gpun::Ciphertext &)>
        eval_func,
    bool async, int32_t n_threads) {
  auto len = cts1.size();
  if (async) {
    ParallelFor(len, n_threads,
                [&](int i) { eval_func(cts1[i], cts2[i], out[i]); });
  } else {
    for (int i = 0; i < len; i++) {
      eval_func(cts1[i], cts2[i], out[i]);
    }
  }
}

void HeKit::EvalInplace(
    std::vector<seal_gpun::Ciphertext> &cts1,
    const std::vector<seal_gpun::Ciphertext> &cts2,
    std::function<void(seal_gpun::Ciphertext &, const seal_gpun::Ciphertext &)>
        eval_func,
    bool async, int32_t n_threads) {
  auto len = cts1.size();
  if (async) {
    ParallelFor(len, n_threads, [&](int i) { eval_func(cts1[i], cts2[i]); });
  } else {
    for (int i = 0; i < len; i++) {
      eval_func(cts1[i], cts2[i]);
    }
  }
}

void HeKit::Add(const seal_gpun::Ciphertext &ct1,
                const seal_gpun::Ciphertext &ct2, seal_gpun::Ciphertext &out) {
  evaluator_->add(ct1, ct2, out);
}

void HeKit::Add(const std::vector<seal_gpun::Ciphertext> &cts1,
                const std::vector<seal_gpun::Ciphertext> &cts2,
                std::vector<seal_gpun::Ciphertext> &out, bool async,
                int32_t n_threads) {
  auto eval_func =
      std::bind(&seal_gpun::Evaluator::add, evaluator_, std::placeholders::_1,
                std::placeholders::_2, std::placeholders::_3);
  Eval(cts1, cts2, out, eval_func, async, n_threads);
}

void HeKit::AddInplace(seal_gpun::Ciphertext &ct1,
                       const seal_gpun::Ciphertext &ct2) {
  evaluator_->addInplace(ct1, ct2);
}

void HeKit::AddInplace(std::vector<seal_gpun::Ciphertext> &cts1,
                       const std::vector<seal_gpun::Ciphertext> &cts2,
                       bool async, int32_t n_threads) {
  auto eval_func = std::bind(&seal_gpun::Evaluator::addInplace, evaluator_,
                             std::placeholders::_1, std::placeholders::_2);
  EvalInplace(cts1, cts2, eval_func, async, n_threads);
}

void HeKit::Sub(const seal_gpun::Ciphertext &ct1,
                const seal_gpun::Ciphertext &ct2, seal_gpun::Ciphertext &out) {
  evaluator_->sub(ct1, ct2, out);
}

void HeKit::Sub(const std::vector<seal_gpun::Ciphertext> &cts1,
                const std::vector<seal_gpun::Ciphertext> &cts2,
                std::vector<seal_gpun::Ciphertext> &out, bool async,
                int32_t n_threads) {
  auto eval_func =
      std::bind(&seal_gpun::Evaluator::sub, evaluator_, std::placeholders::_1,
                std::placeholders::_2, std::placeholders::_3);
  Eval(cts1, cts2, out, eval_func, async, n_threads);
}

void HeKit::SubInplace(seal_gpun::Ciphertext &ct1,
                       const seal_gpun::Ciphertext &ct2) {
  evaluator_->subInplace(ct1, ct2);
}

void HeKit::SubInplace(std::vector<seal_gpun::Ciphertext> &cts1,
                       const std::vector<seal_gpun::Ciphertext> &cts2,
                       bool async, int32_t n_threads) {
  auto eval_func = std::bind(&seal_gpun::Evaluator::subInplace, evaluator_,
                             std::placeholders::_1, std::placeholders::_2);
  EvalInplace(cts1, cts2, eval_func, async, n_threads);
}

void HeKit::Multiply(const seal_gpun::Ciphertext &ct1,
                     const seal_gpun::Ciphertext &ct2,
                     seal_gpun::Ciphertext &out) {
  out = ct1;
  MultiplyInplace(out, ct2);
}

void HeKit::MultiplyInplace(seal_gpun::Ciphertext &ct1,
                            const seal_gpun::Ciphertext &ct2) {
  evaluator_->multiplyInplace(ct1, ct2);
  // evaluator_->relinearizeInplace(ct1, rlk_);
  // evaluator_->rescaleToNextInplace(ct1);
}

void HeKit::MultiplyPlain(const seal_gpun::Ciphertext &ct,
                          const seal_gpun::Plaintext &pt,
                          seal_gpun::Ciphertext &out) {
  out = ct;
  MultiplyPlainInplace(out, pt);
}

void HeKit::MultiplyPlainInplace(seal_gpun::Ciphertext &ct,
                                 const seal_gpun::Plaintext &pt) {
  evaluator_->multiplyPlainInplace(ct, pt);
  evaluator_->relinearizeInplace(ct, rlk_);
  evaluator_->rescaleToNextInplace(ct);
}

void HeKit::RotateSum(seal_gpun::Ciphertext &ct) {
  seal_gpun::Ciphertext rotated;
  for (int i = 0; i < log2(poly_modulus_degree_) - 1; ++i) {
    evaluator_->rotateVector(ct, pow(2, i), glk_, rotated);
    evaluator_->addInplace(ct, rotated);
  }
}

//==========================fhe_gpu operation end============================

//==========================fhe_gpu template begin===========================
template void HeKit::Encode(const std::vector<int64_t> &ms,
                            std::vector<seal_gpun::Plaintext> &out, bool async,
                            int32_t n_threads);
template void HeKit::Encode(const std::vector<double> &ms,
                            std::vector<seal_gpun::Plaintext> &out, bool async,
                            int32_t n_threads);

template void HeKit::Encrypt(const int64_t m, seal_gpun::Ciphertext &out);
template void HeKit::Encrypt(const double m, seal_gpun::Ciphertext &out);

template void HeKit::Encrypt(const std::vector<int64_t> &pts,
                             std::vector<seal_gpun::Ciphertext> &out,
                             bool async, int32_t n_threads);
template void HeKit::Encrypt(const std::vector<double> &pts,
                             std::vector<seal_gpun::Ciphertext> &out,
                             bool async, int32_t n_threads);
template void HeKit::Encrypt(const std::vector<seal_gpun::Plaintext> &pts,
                             std::vector<seal_gpun::Ciphertext> &out,
                             bool async, int32_t n_threads);
//==========================fhe_gpu template end=============================
}  // namespace heu::algos::seal_fhe::gpu
